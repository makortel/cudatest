// -*- c++ -*-
// nvcc -o transfer transfer.cu

#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <random>
#include <tuple>
#include <thread>
#include <mutex>
#include <cassert>

constexpr size_t MAXTHREADS = 8;
constexpr size_t TIMES = 10;
constexpr size_t MAX = 1<<27; // 128Melements = 512 MB
constexpr size_t MAXOPS = 1000000;

//#define OWN_GLOBAL_MUTEX
//#define OWN_PER_CALL_MUTEX

class Data;
void transfer(Data *data, const size_t opsPerLock);

std::mutex cudaMutex;

struct Data {
  Data() {
    hipStreamCreate(&stream);

    hipMalloc(&a_d, MAX*sizeof(float));
    hipHostMalloc(&a_h, MAX*sizeof(float), hipHostMallocDefault);

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(1e-5, 100.);
    for(size_t i=0; i<MAX; ++i) {
      a_h[i] = dis(gen);
    }
  }
  ~Data() {
    hipStreamSynchronize(stream);
    hipHostFree(a_h);
    hipFree(a_d);
    hipStreamDestroy(stream);
  }

  void transferAsync(size_t opsPerLock) {
    thread = std::thread{transfer, this, opsPerLock};
  }

  double wait() {
    thread.join();
    return time;
  }

  std::thread thread;
  hipStream_t stream;
  double time;
  float *a_d;
  float *a_h;
};

void transfer(Data *data, const size_t opsPerLock) {
  assert(MAXOPS % opsPerLock == 0);
  auto start = std::chrono::high_resolution_clock::now();

#ifdef OWN_GLOBAL_MUTEX
  std::lock_guard<std::mutex> lock{cudaMutex};
#endif

  for(size_t i=0, j=0; i<MAXOPS/opsPerLock; ++i) {
#ifdef OWN_PER_CALL_MUTEX
    std::lock_guard<std::mutex> lock{cudaMutex};
#endif
    for(size_t k=0; k<opsPerLock; ++k) {
      hipMemcpyAsync(data->a_d+j, data->a_h+j, sizeof(float), hipMemcpyDefault, data->stream);
      j = (j+1)%MAX;
    }
  }

  auto stop = std::chrono::high_resolution_clock::now();
  data->time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(stop-start).count())/1e6;
}

int main() {
  std::vector<Data> threads(MAXTHREADS);

  for(size_t nth=1; nth<=MAXTHREADS; ++nth) {
    std::cout << "Number of threads " << nth << std::endl;
    double total = 0;
    for(size_t i=0; i<TIMES; ++i) {
      std::cout << "Trial " << i << std::endl;
      for(size_t j=0; j<nth; ++j) {
        threads[j].transferAsync(1);
      }
      for(size_t j=0; j<nth; ++j) {
        total += threads[j].wait();
      }
    }
    total = total / TIMES;
    std::cout << "Ops " << (MAXOPS*nth) << " time/trial " << total << " ops/s " << (MAXOPS*nth/total) << " us/op " << (total/(MAXOPS*nth)*1e6) << std::endl;
  }

#ifdef OWN_PER_CALL_MUTEX
  std::cout << "Ops per lock for " << MAXTHREADS << " threads " << std::endl;
  for(size_t opsPerLock=2; opsPerLock <= 64; opsPerLock = opsPerLock << 1) {
    double total = 0;
    for(size_t i=0; i<TIMES; ++i) {
      std::cout << "Trial " << i << std::endl;
      for(auto& th: threads) {
        th.transferAsync(opsPerLock);
      }
      for(auto& th: threads) {
        total += th.wait();
      }
    }
    total = total / TIMES;
    std::cout << "Ops " << (MAXOPS*MAXTHREADS) << " per lock " << opsPerLock << " time/trial " << total << " ops/s " << (MAXOPS*MAXTHREADS/total) << " us/op " << (total/(MAXOPS*MAXTHREADS)*1e6) << std::endl;
  }
#endif

  return 0;
}
