// -*- c++ -*-
// nvcc -std=c++14 -o callback2 callback2.cu

// as callback4 but with multiple streams per thread


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <atomic>
#include <vector>
#include <thread>

__global__ void kernel()
{
   int idx = threadIdx.x + blockIdx.x * blockDim.x;
   int stride = gridDim.x * blockDim.x;
   volatile float x = 0;

   for (int j = idx; j < 1024*1024*64; j += stride) {
       #pragma unroll
       for (int i = 0; i < 512; ++i) {
           x += float(i)*float(i);
       }
   }
}

constexpr size_t NSTREAMS = 32;
constexpr size_t NTHREADS = 1;
constexpr size_t STREAMS_PER_THREAD = NSTREAMS/NTHREADS;
constexpr int numThreads = 256;
constexpr int numBlocks = 20;

//#define CREATE_THREADS
//#define ADD_CALLBACK

std::atomic<bool> canStart;
std::atomic<bool> canContinue[NSTREAMS];
std::atomic<int> waiting;

void CUDART_CB cb(hipStream_t stream, hipError_t status, void *data) {
  canContinue[reinterpret_cast<size_t>(data)].store(true);
  --waiting;
}

int main(int argc, char* argv[])
{
   hipStream_t streams[NSTREAMS];

   canStart = false;
   for (int i = 0; i < NSTREAMS; ++i) {
       hipStreamCreate(&streams[i]);
       canContinue[i].store(false);
   }

   hipDeviceSynchronize();

   waiting.store(NSTREAMS);
#ifdef CREATE_THREADS
   std::vector<std::thread> threads(NTHREADS);
   for (size_t i = 0; i < NTHREADS; ++i) {
     threads[i] = std::thread{[ith=i, &streams]() {
         while(not canStart.load()) {}
#else
         const size_t ith = 0;
#endif // CREATE_THREADS

         for (int j = 0; j < 4; ++j) {
           for(size_t k = 0; k<STREAMS_PER_THREAD; ++k) {
             const size_t ist = ith*STREAMS_PER_THREAD+k;
             kernel<<<numBlocks, numThreads, 0, streams[ist]>>>();
             //kernel<<<numBlocks, numThreads, 0, streams[ist]>>>();
#ifdef ADD_CALLBACK
             if (j == 2) {
               cudaStreamAddCallback(streams[ist], cb, (void*) ist, 0);
             }
#endif // ADD_CALLBACK
           }
#ifdef ADD_CALLBACK
           if(j == 2) {
             for(size_t k = 0; k<STREAMS_PER_THREAD; ++k) {
               const size_t ist = ith*STREAMS_PER_THREAD+k;
               while(not canContinue[ist].load()) {}
               while(not waiting.load() == 0) {}
             }
           }
#endif // ADD_CALLBACK
         }

         for(size_t k = 0; k<STREAMS_PER_THREAD; ++k) {
           const size_t ist = ith*STREAMS_PER_THREAD+k;
           hipStreamSynchronize(streams[ist]);
         }
#ifdef CREATE_THREADS
       }};
   }
   canStart.store(true);

   for(auto& th: threads) {
     th.join();
   }
#endif // CREATE_THREADS

   hipDeviceSynchronize();

   for (int i = 0; i < NSTREAMS; ++i) {
       hipStreamDestroy(streams[i]);
   }

   return 0;
}
