// -*- c++ -*-
// nvcc -std=c++14 -o callback2 callback2.cu

// as callback4 but with multiple streams per thread


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <atomic>
#include <vector>
#include <thread>

__global__ void kernel()
{
   int idx = threadIdx.x + blockIdx.x * blockDim.x;
   int stride = gridDim.x * blockDim.x;
   volatile float x = 0;

   for (int j = idx; j < 1024*1024*16; j += stride) {
       #pragma unroll
       for (int i = 0; i < 128; ++i) {
           x += float(i)*float(i);
       }
   }
}

constexpr size_t NSTREAMS = 32;
constexpr size_t NTHREADS = 4;
constexpr size_t STREAMS_PER_THREAD = NSTREAMS/NTHREADS;
constexpr int numThreads = 256;
constexpr int numBlocks = 20;

std::atomic<bool> canStart;
std::atomic<bool> canContinue[NSTREAMS];
std::atomic<int> waiting;

void CUDART_CB cb(hipStream_t stream, hipError_t status, void *data) {
  canContinue[reinterpret_cast<size_t>(data)].store(true);
  --waiting;
}

int main(int argc, char* argv[])
{
   hipStream_t streams[NSTREAMS];

   canStart = false;
   for (int i = 0; i < NSTREAMS; ++i) {
       hipStreamCreate(&streams[i]);
       canContinue[i].store(false);
   }

   hipDeviceSynchronize();

   std::vector<std::thread> threads(NTHREADS);
   waiting.store(NSTREAMS);
   for (size_t i = 0; i < NTHREADS; ++i) {
     threads[i] = std::thread{[ith=i, &streams]() {
         while(not canStart.load()) {}

         for (int j = 0; j < 4; ++j) {
           for(size_t k = 0; k<STREAMS_PER_THREAD; ++k) {
             const size_t ist = ith*STREAMS_PER_THREAD+k;
             kernel<<<numBlocks, numThreads, 0, streams[ist]>>>();
             kernel<<<numBlocks, numThreads, 0, streams[ist]>>>();
             if (j == 2) {
               hipStreamAddCallback(streams[ist], cb, (void*) ist, 0);
             }
           }
           if(j == 2) {
             for(size_t k = 0; k<STREAMS_PER_THREAD; ++k) {
               const size_t ist = ith*STREAMS_PER_THREAD+k;
               while(not canContinue[ist].load()) {}
               while(not waiting.load() == 0) {}
             }
           }
         }
       }};
   }
   canStart.store(true);

   for(auto& th: threads) {
     th.join();
   }

   hipDeviceSynchronize();

   for (int i = 0; i < NSTREAMS; ++i) {
       hipStreamDestroy(streams[i]);
   }

   return 0;
}
