// -*- c++ -*-
// nvcc -std=c++14 -o callback2 callback2.cu

// as callback3 but with threads


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <atomic>
#include <vector>
#include <thread>

__global__ void kernel()
{
   int idx = threadIdx.x + blockIdx.x * blockDim.x;
   int stride = gridDim.x * blockDim.x;
   volatile float x = 0;

   for (int j = idx; j < 1024*1024*64; j += stride) {
       #pragma unroll
       for (int i = 0; i < 512; ++i) {
           x += float(i)*float(i);
       }
   }
}

constexpr size_t NSTREAMS = 16;
//constexpr int numThreads = 256;
constexpr int numThreads = 32;
constexpr int numBlocks = 1;

std::atomic<bool> canStart;
std::atomic<bool> canContinue[NSTREAMS];
std::atomic<int> waiting;

void CUDART_CB cb(hipStream_t stream, hipError_t status, void *data) {
  canContinue[reinterpret_cast<size_t>(data)].store(true);
  --waiting;
}

int main(int argc, char* argv[])
{
   hipStream_t streams[NSTREAMS];

   canStart = false;
   for (int i = 0; i < NSTREAMS; ++i) {
       hipStreamCreate(&streams[i]);
       canContinue[i].store(false);
   }

   hipDeviceSynchronize();

   std::vector<std::thread> threads(NSTREAMS);
   waiting.store(NSTREAMS);
   for (size_t i = 0; i < NSTREAMS; ++i) {
     threads[i] = std::thread{[index=i, &streams]() {
         while(not canStart.load()) {}

         for (int j = 0; j < 4; ++j) {
           kernel<<<numBlocks, numThreads, 0, streams[index]>>>();
           kernel<<<numBlocks, numThreads, 0, streams[index]>>>();
           if (j == 2) {
             hipStreamAddCallback(streams[index], cb, (void*) index, 0);
             while(not canContinue[index].load()) {}
             while(not waiting.load() == 0) {}
           }
         }
       }};
   }
   canStart.store(true);

   for(auto& th: threads) {
     th.join();
   }

   hipDeviceSynchronize();

   for (int i = 0; i < NSTREAMS; ++i) {
       hipStreamDestroy(streams[i]);
   }

   return 0;
}
