// -*- c++ -*-
// nvcc -o transfer transfer.cu

#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <random>
#include <tuple>
#include <thread>
#include <mutex>
#include <cassert>

constexpr size_t STREAMS = 64;
//constexpr size_t STREAMS = 256;
//constexpr size_t STREAMS = 50000;
constexpr size_t CUDATHREADS = 32; // 1 warp
constexpr size_t ELEMENTS = STREAMS*CUDATHREADS;
constexpr size_t LOOPS = 1000000;

__global__ void kernel_looping(float *a) {
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  for(size_t iloop=0; iloop<LOOPS; ++iloop) {
    a[idx] = a[idx] + 1.0f;
  }
}

int main() {
  std::vector<hipStream_t> streams(STREAMS);
  for(auto& s: streams) {
    hipStreamCreate(&s);
  }

  float *data_d;
  float *data_h;
  hipMalloc(&data_d, ELEMENTS*sizeof(float));
  hipHostMalloc(&data_h, ELEMENTS*sizeof(float), hipHostMallocDefault);

  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<float> dis(1e-5, 100.);
  for(size_t i=0; i<ELEMENTS; ++i) {
    data_h[i] = dis(gen);
  }
  hipMemcpyAsync(data_d, data_h, ELEMENTS*sizeof(float), hipMemcpyDefault, streams[0]);
  hipStreamSynchronize(streams[0]);

  for(size_t i=0; i<STREAMS; ++i) {
    kernel_looping<<<1, CUDATHREADS, 0, streams[i]>>>(data_d+i*CUDATHREADS);
  }

  for(auto& s: streams) {
    hipStreamSynchronize(s);
  }

  for(auto& s: streams) {
    hipStreamDestroy(s);
  }

  hipHostFree(data_h);
  hipFree(data_d);

  return 0;
}
